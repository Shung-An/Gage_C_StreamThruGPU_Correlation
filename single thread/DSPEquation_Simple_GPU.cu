#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>
#include <stdlib.h>
#include <time.h>

#define DIM 768

#define SMEMDIM 100 
#define N 1000704


__global__ void plusOne(unsigned char *a, __int64 numElements, unsigned long skip)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		unsigned char temp = a[i] + 1;
		int index = i / skip;
		if (i % skip == 0)
			a[index] = temp;
		//printf("%hhu\n", a[i]);
	}
}

// Cast a data to a double and use the window data if it exists
__global__ void byteToDouble(unsigned char* in, double* window, double* out, __int64 numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		if (window)
		{
			out[i] = (double)in[i] * window[i];
		}
		else
		{
			out[i] = (double)in[i];
		}
	}
}



__global__ void plusOneShort1(short* a, __int64 numElements, unsigned long skip)
{
	__shared__ int smem[SMEMDIM];
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
#pragma unroll
	for (int i = index; i < numElements / 4; i += stride)
	{
		a[i * 4] = (a[i * 4] - a[i * 4 + 2]);
		a[i * 4 + 1] = (a[i * 4 + 1] - a[i * 4 + 3]);
	}

}

__global__ void plusOneShort2(short* a, __int64 numElements, int* out)
{
	__shared__ int smem[SMEMDIM];
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
#pragma unroll
	for (int i = index; i < numElements / 4; i += stride)
	{
		int a1 = a[i * 4];
		int a2 = a[i * 4 + 1];
		int a3 = a[i * 4 + 2];
		int a4 = a[i * 4 + 3];
		int temp1;
		int temp2;
		temp1 = a1 - a3;
		temp2 = a2 - a4;
		out[i] = temp1 * temp2;
	}

}

__global__ void multiplyKernel(short* a, int* dev_a,  __int64 numElements)
{
	__shared__ int smem[SMEMDIM];
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
#pragma unroll
	for (int i = index; i < numElements /4; i+=stride)
	{
		dev_a[i] = (int)a[i * 4 ] * (int)a[i * 4 + 1];
	}

}

__inline__ __device__ int warpReduce(int mySum) {
	mySum += __shfl_xor(mySum, 16);
	mySum += __shfl_xor(mySum, 8);
	mySum += __shfl_xor(mySum, 4);
	mySum += __shfl_xor(mySum, 2);
	mySum += __shfl_xor(mySum, 1);
	return mySum;
}

__global__ void subtraction(int* a, int* out, unsigned int n) {
	__shared__ int smem[SMEMDIM];
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;
#pragma unroll
	for (int i = idx;  i < n/4; i += stride)
	{
		int a1 = a[i*4];
		int a2 = a[i*4 + 1];
		int a3 = a[i*4+2];	
		int a4 = a[i*4 + 3];
		int temp1;
		int temp2;
		temp1 = a1 - a2;
		temp2 = a3 - a4;
		out[i] = temp1*temp2;
	}
}


__global__ void reduceShfl(int* g_idata, int* g_odata,
	unsigned int n)
{
	// shared memory for each warp sum
	__shared__ int smem[SMEMDIM];

	// boundary check   
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n) return;

	// read from global memory
	int mySum = g_idata[idx];

	// calculate lane index and warp index
	int laneIdx = threadIdx.x % warpSize;
	int warpIdx = threadIdx.x / warpSize;

	// block-wide warp reduce 
	mySum = warpReduce(mySum);

	// save warp sum to shared memory
	if (laneIdx == 0) smem[warpIdx] = mySum;

	// block synchronization
	__syncthreads();

	// last warp reduce
	mySum = (threadIdx.x < SMEMDIM) ? smem[laneIdx] : 0;
	if (warpIdx == 0) mySum = warpReduce(mySum);

	// write result for this block to global mem
	if (threadIdx.x == 0) atomicAdd(g_odata,mySum);
}
__global__ void initializeArray(int* array) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N) {
		array[idx] = 1;
	}
}

__global__ void resetInteger(int* value) {
	*value = 0; // Reset integer value
}

// Helper function for using CUDA.

extern "C" hipError_t GPU_Equation_PlusOne(void* a, unsigned long skip, unsigned long sample_size, __int64 size, int blocks, int threads, int u32LoopCount, int* h_odata, short* h_dev_a, short* h_dev_a2, int * dev_a, void* d_accTemp, void * d_accTemp2)
{
	hipError_t cudaStatus = hipSuccess;

	blocks = 48 * 32;
	threads = 768;
	clock_t start_Time, current_time;
	double elapsed_time;

	int CPUresult = 1 ;
	int CheckRaw = 0;
	int AnalysisFile = 1;

	int h_accTemp2 = 0;

	FILE* fptr;
	if (AnalysisFile == 1) {
		fptr = fopen("Analysis.txt", "a");
	}
	//start_Time = clock();

	cudaStatus = hipMemcpy(h_dev_a, a, size * sizeof(short), hipMemcpyDeviceToHost);

	plusOneShort2 << <blocks, threads >> > ((short*)a, size, dev_a);
	reduceShfl << <blocks, threads >> > (dev_a, (int*) d_accTemp2, size/4);
	hipMemcpy(h_odata, d_accTemp2, 1 * sizeof(int), hipMemcpyDeviceToHost);
	resetInteger << <1,1 >> > ((int*)d_accTemp2);

	cudaStatus = hipMemcpy(h_dev_a2, a, size * sizeof(short), hipMemcpyDeviceToHost);
	cudaStatus = hipDeviceSynchronize();
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.


	if (CPUresult == 1) {
		for (int i = 0; i < size; i++) {
			if (i % 4 == 0 && i < size - 2)
			{
				//// Write to local disk
				if (1 == CheckRaw) {
					if (h_dev_a[i] - h_dev_a[i + 2] != h_dev_a2[i])
					{
						h_dev_a[i] = h_dev_a[i] - h_dev_a[i + 2];
						h_dev_a[i + 1] = h_dev_a[i + 1] - h_dev_a[i + 3];
						printf("%d\t%d\t%d\t%d\t%d\t%d\t%d\t%d\t%d\n", i, h_dev_a2[i], h_dev_a[i], h_dev_a2[i + 1], h_dev_a[i + 1], h_dev_a2[i + 2], h_dev_a[i + 2], h_dev_a2[i + 3], h_dev_a[i + 3]);
						//h_dev_a[i] = h_dev_a[i] * h_dev_a[i + 1];
						////printf("%d\t%d\t%d\n", i, h_dev_a[i], h_dev_a2[i / 4]);
						//h_dev_a[i + 1] = 0;
					}
				}
				else {
					h_dev_a[i] = h_dev_a[i] - h_dev_a[i + 2];
					h_dev_a[i + 1] = h_dev_a[i + 1] - h_dev_a[i + 3];
					h_dev_a[i + 2] = 0;
					h_dev_a[i + 3] = 0;
					h_dev_a[i] = h_dev_a[i] * h_dev_a[i + 1];
					h_dev_a[i + 1] = 0;
					h_accTemp2 += h_dev_a[i];
				}
			}
		}
	}


	if (CheckRaw != 1) {
		if (CPUresult == 1) {
			if (AnalysisFile == 1) {
				fprintf(fptr, "%d\t%d\t%d\n", u32LoopCount,h_accTemp2, h_odata[0]);
				fclose(fptr);
			}
		}
		else {
			if (AnalysisFile == 1) {
				fprintf(fptr, "%d\t%lld\n", u32LoopCount, h_odata[0]);
				fclose(fptr);
			}
		}
	}


	fclose(fptr);
	// Get the current time
	//current_time = clock();
	//elapsed_time = ((double)(current_time - start_Time)) / CLOCKS_PER_SEC * 1000;
	//printf("Elapsed Time: %.2f ms\r", elapsed_time);


	return cudaStatus;
}
